#include "hip/hip_runtime.h"

#include <stdio.h>
#include <iostream>

using namespace std;


__global__ void 
add(float *g, float *o, const int dimx, const int dimy, const int add) {

	//extern __shared__ float sdata[];

	//unsigned int tid_x = threadIdx.x;
	//unsigned int tid_y = threadIdx.y;

	unsigned int i = blockDim.x * blockIdx.x + threadIdx.x;
	unsigned int j = blockDim.y * blockIdx.y + threadIdx.y; 

	if (i >= dimx || j >= dimy)
	    return;

	o[i*dimy+j] = g[i*dimy+j] + add;
}



int
main()
{   
	int dimx = 32;
	int dimy = 16;
	int num_bytes = dimx*dimy*sizeof(float);
	int add = 3;

	float *d_a, *h_a, // device and host pointers
	            *d_o, *h_o;

	h_a = (float*)malloc(num_bytes);
	h_o = (float*)malloc(num_bytes);

	for (int i=0; i < dimx; i++){   
	    for (int j=0; j < dimy; j++){
	        h_a[i*dimy + j] = 1;
	    }
	}
	for (int i=0; i < dimx; i++){   
	    for (int j=0; j < dimy; j++){
	        h_o[i*dimy + j] = 0;
	    }
	}

	hipMalloc( (void**)&d_a, num_bytes );
	hipMalloc( (void**)&d_o, num_bytes );

	hipMemcpy( d_a, h_a, num_bytes, hipMemcpyHostToDevice);
	hipMemcpy( d_o, h_o, num_bytes, hipMemcpyHostToDevice); 

	dim3 grid, block;
	block.x = 4;
	block.y = 4;
	grid.x = dimx / block.x;
	grid.y = dimy / block.y;

	add<<<grid, block>>> (d_a, d_o, dimx, dimy, add);

	std::cout << block.x << " " << block.y << std::endl;
	std::cout << grid.x << " " << grid.y << std::endl;
	std::cout << dimx <<  " " << dimy << " " << dimx*dimy << std::endl;

	hipMemcpy( h_a, d_a, num_bytes, hipMemcpyDeviceToHost );
	hipMemcpy( h_o, d_o, num_bytes, hipMemcpyDeviceToHost );

	hipFree(d_a);
	hipFree(d_o);

	printf("MATRIX A\n\t");
    int row, col;
	for (row = 0; row < dimx; row++) {
      for (col = 0; col < dimy; col++) {
          printf("%1.0f%s", h_a[(row*dimy+col)], (col < dimy-1) ? ", " : ";\n\t");
      }
    } 

	printf("MATRIX O\n\t");
  	for (row = 0; row < dimx; row++) {
      for (col = 0; col < dimy; col++) {
          printf("%1.0f%s", h_o[(row*dimy+col)], (col < dimy-1) ? ", " : ";\n\t");
      }
    } 
    free(h_a);
    free(h_o);
}
