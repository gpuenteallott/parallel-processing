// This is the REAL "hello world" for CUDA!
// It takes the string "Hello ", prints it, then passes it to CUDA with an array
// of offsets. Then the offsets are added in parallel to produce the string "World!"
// By Ingemar Ragnemalm 2010
// http://computer-graphics.se/hello-world-for-cuda.html
 

#include <hip/hip_runtime.h>
#include <stdio.h>
 
const int N = 16; 
const int blocksize = 16; 
 
__global__ 
void hello(char *a, int *b) 
{
	a[threadIdx.x] += b[threadIdx.x];
}
 
int main()
{
	char a[N] = "Hello \0\0\0\0\0\0";
	int b[N] = {15, 10, 6, 0, -11, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0};

	float A[N][N];
	float dA[N][N];
 
	char *ad;
	int *bd;
	const int csize = N*sizeof(char);
	const int isize = N*sizeof(int);
	const int asize = N*N*sizeof(float);
 
	printf("%s", a);
 
	hipMalloc( (void**)&ad, csize ); 
	hipMalloc( (void**)&bd, isize ); 
	hipMalloc( (void**)&dA, asize ); 
	hipMemcpy( ad, a, csize, hipMemcpyHostToDevice ); 
	hipMemcpy( bd, b, isize, hipMemcpyHostToDevice ); 
	hipMemcpy( dA, A, asize, hipMemcpyHostToDevice ); 
	
	dim3 dimBlock( blocksize, 1 );
	dim3 dimGrid( 1, 1 );
	hello<<<dimGrid, dimBlock>>>(ad, bd);
	hipMemcpy( a, ad, csize, hipMemcpyDeviceToHost ); 
	hipMemcpy( A, dA, asize, hipMemcpyDeviceToHost ); 
	hipFree( ad );
	hipFree( bd );
	hipFree( dA );

	int row, col;
	for (row = 0; row < N; row++) {
        for (col = 0; col < N; col++) {
            printf("%1.10f%s", A[row][col], (col < N-1) ? ", " : ";\n\t");
        }
    }
	
	printf("%s\n", a);
	return EXIT_SUCCESS;
}