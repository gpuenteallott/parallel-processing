#include "hip/hip_runtime.h"
// This is the REAL "hello world" for CUDA!
// It takes the string "Hello ", prints it, then passes it to CUDA with an array
// of offsets. Then the offsets are added in parallel to produce the string "World!"
// By Ingemar Ragnemalm 2010
// http://computer-graphics.se/hello-world-for-cuda.html
 
#include <stdio.h>
 
const int N = 16; 
const int blocksize = 16; 
 
__global__ 
void hello(float **dA) 
{
	dA[threadIdx.x][threadIdx.y] = threadIdx.x;
}
 
int main()
{
	char a[N] = "Hello \0\0\0\0\0\0";
	int b[N] = {15, 10, 6, 0, -11, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0};

	float A[N][N];
	float dA[N][N];
 
	const int asize = N*N*sizeof(float);
 
 
	hipMalloc( (void**)&dA, asize ); 
	hipMemcpy( dA, A, asize, hipMemcpyHostToDevice ); 
	
	dim3 dimBlock( blocksize, blocksize );
	dim3 dimGrid( 1, 1 );
	hello<<<dimGrid, dimBlock>>>(&dA);
	hipMemcpy( A, dA, asize, hipMemcpyDeviceToHost ); 
	hipFree( dA );

	int row, col;
	for (row = 0; row < N; row++) {
        for (col = 0; col < N; col++) {
            printf("%1.10f%s", A[row][col], (col < N-1) ? ", " : ";\n\t");
        }
    }
	
	return EXIT_SUCCESS;
}