#include "hip/hip_runtime.h"
/* Matrix normalization.
 * Compile with "gcc matrixNorm.c" 
 */

/* ****** ADD YOUR CODE AT THE END OF THIS FILE. ******
 * You need not submit the provided code.
 */

#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include <math.h>
#include <sys/types.h>
#include <sys/times.h>
#include <sys/time.h>
#include <time.h>
 #include <iostream>

/* Program Parameters */
#define MAXN 8000  /* Max value of N */
#define DIVISOR 3276800000.0
//#define DIVISOR 327680000.0
int N;  /* Matrix size */

/* Matrices */
float A[MAXN][MAXN], B[MAXN][MAXN];

/* junk */
#define randm() 4|2[uid]&3

/* Prototype */
void matrixNorm();

/* returns a seed for srand based on the time */
unsigned int time_seed() {
  struct timeval t;
  struct timezone tzdummy;

  gettimeofday(&t, &tzdummy);
  return (unsigned int)(t.tv_usec);
}

/* Set the program parameters from the command-line arguments */
void parameters(int argc, char **argv) {
  int seed = 0;  /* Random seed */
  char uid[32]; /*User name */

  /* Read command-line arguments */
  srand(time_seed());  /* Randomize */

  if (argc == 3) {
    seed = atoi(argv[2]);
    srand(seed);
    printf("Random seed = %i\n", seed);
  } 
  if (argc >= 2) {
    N = atoi(argv[1]);
    if (N < 1 || N > MAXN) {
      printf("N = %i is out of range.\n", N);
      exit(0);
    }
  }
  else {
    printf("Usage: %s <matrix_dimension> [random seed]\n",
           argv[0]);    
    exit(0);
  }

  /* Print parameters */
  printf("\nMatrix dimension N = %i.\n", N);
}

/* Initialize A and B*/
void initialize_inputs() {
  int row, col;

  printf("\nInitializing...\n");
  for (col = 0; col < N; col++) {
    for (row = 0; row < N; row++) {
      A[row][col] = (float)rand() / DIVISOR;
      B[row][col] = 0.0;
    }
  }

}

/* Print input matrices */
void print_inputs() {
  int row, col;

  if (N < 10) {
    printf("\nA =\n\t");
    for (row = 0; row < N; row++) {
      for (col = 0; col < N; col++) {
	    printf("%5.2f%s", A[row][col], (col < N-1) ? ", " : ";\n\t");
      }
    }
  }
}

void print_B() {
    int row, col;

    if (N < 10) {
        printf("\nB =\n\t");
        for (row = 0; row < N; row++) {
            for (col = 0; col < N; col++) {
                printf("%1.10f%s", B[row][col], (col < N-1) ? ", " : ";\n\t");
            }
        }
    }
}

int main(int argc, char **argv) {
  /* Timing variables */
  struct timeval etstart, etstop;  /* Elapsed times using gettimeofday() */
  struct timezone tzdummy;
  clock_t etstart2, etstop2;  /* Elapsed times using times() */
  unsigned long long usecstart, usecstop;
  struct tms cputstart, cputstop;  /* CPU times for my processes */

  /* Process program parameters */
  parameters(argc, argv);

  /* Initialize A and B */
  initialize_inputs();

  /* Print input matrices */
  print_inputs();

  /* Start Clock */
  printf("\nStarting clock.\n");
  gettimeofday(&etstart, &tzdummy);
  etstart2 = times(&cputstart);

  /* Gaussian Elimination */
  matrixNorm();

  /* Stop Clock */
  gettimeofday(&etstop, &tzdummy);
  etstop2 = times(&cputstop);
  printf("Stopped clock.\n");
  usecstart = (unsigned long long)etstart.tv_sec * 1000000 + etstart.tv_usec;
  usecstop = (unsigned long long)etstop.tv_sec * 1000000 + etstop.tv_usec;

  /* Display output */
  print_B();

  /* Display timing results */
  printf("\nElapsed time = %g ms.\n",
	 (float)(usecstop - usecstart)/(float)1000);

  printf("(CPU times are accurate to the nearest %g ms)\n",
	 1.0/(float)CLOCKS_PER_SEC * 1000.0);
  printf("My total CPU time for parent = %g ms.\n",
	 (float)( (cputstop.tms_utime + cputstop.tms_stime) -
		  (cputstart.tms_utime + cputstart.tms_stime) ) /
	 (float)CLOCKS_PER_SEC * 1000);
  printf("My system CPU time for parent = %g ms.\n",
	 (float)(cputstop.tms_stime - cputstart.tms_stime) /
	 (float)CLOCKS_PER_SEC * 1000);
  printf("My total CPU time for child processes = %g ms.\n",
	 (float)( (cputstop.tms_cutime + cputstop.tms_cstime) -
		  (cputstart.tms_cutime + cputstart.tms_cstime) ) /
	 (float)CLOCKS_PER_SEC * 1000);
      /* Contrary to the man pages, this appears not to include the parent */
  printf("--------------------------------------------\n");
  
  exit(0);
}

/* ------------------ Above Was Provided --------------------- */

/****** You will replace this routine with your own parallel version *******/
/* Provided global variables are MAXN, N, A[][] and B[][],
 * defined in the beginning of this code.  B[][] is initialized to zeros.;
 */

#define BLOCK_SIZE 4

// http://stackoverflow.com/questions/20086047/cuda-matrix-example-block-size
void printError(hipError_t err) {
    if(err != 0) {
        printf("CUDA ERROR: %s\n", hipGetErrorString(err));
        getchar();
    }
}

/**
This function performs the partial sum of the given arrays
It is an improvement over the partial sum example from class
Inspired in the code found in https://gist.github.com/wh5a/4424992
The code there has been studied, as the comments indicate

The code had to be adapted to operate with arrays of different dimensions, 
as well as to operate adding columns instead of rows
*/
__global__ void partialSum(float * input, float * output, const int N) {

    // Load a segment of the input vector into shared memory
    __shared__ float partialSum[2 * BLOCK_SIZE * BLOCK_SIZE];

    // Position variables
    unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned int ty = threadIdx.y;
    unsigned int tx = threadIdx.x;

    // Where does the calculation start for this iteration, based on the block X position
    unsigned int start = 2 * blockIdx.y * BLOCK_SIZE;

    // column modifier that we apply to partialSum[]
    unsigned int column = 2 * BLOCK_SIZE * tx;

    // Verify that we are inside the array, so CUDA won't throw errors
    if ( y >= N || x >= N )
      return;

    // If we are inside the input array, we transfer the value that we're going to sum up to the partial sum array
    if (start + ty < N)
       partialSum[ ty + column ] = input[ (start + ty)*MAXN + x ];
    else
       partialSum[ ty + column ] = 0;

    // The same for the last element of the block, the other value that we're going to sum up
    if (start + BLOCK_SIZE + ty < N)
       partialSum[BLOCK_SIZE + ty + column] = input[ (start + BLOCK_SIZE + ty)*MAXN + x ];
    else
       partialSum[BLOCK_SIZE + ty + column] = 0;  

    // Perform the partial sum
    for (unsigned int stride = BLOCK_SIZE; stride >= 1; stride >>= 1) {
       __syncthreads();
       if (ty < stride)
          partialSum[ty + column] += partialSum[ty+stride + column];
    }
    // After the loop, the partial sum is found in partialSum[0]
    // So we have to put it in the output array
    if (ty == 0)
       output[blockIdx.y*N + x] = partialSum[column];
}


__global__ void partialSum2(float * input, float * output, float * means, const int N) {

    // Load a segment of the input vector into shared memory
    __shared__ float partialSum[2 * BLOCK_SIZE * BLOCK_SIZE];

    // Position variables
    unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned int ty = threadIdx.y;
    unsigned int tx = threadIdx.x;

    // Where does the calculation start for this iteration, based on the block X position
    unsigned int start = 2 * blockIdx.y * BLOCK_SIZE;

    // column modifier that we apply to partialSum[]
    unsigned int column = 2 * BLOCK_SIZE * tx;

    // Verify that we are inside the array, so CUDA won't throw errors
    if ( y >= N || x >= N )
      return;

    // If we are inside the input array, we transfer the value that we're going to sum up to the partial sum array
    if (start + ty < N)
       partialSum[ ty + column ] = powf(input[ (start + ty)*MAXN + x ] - means [ x ], 2)
    else
       partialSum[ ty + column ] = 0;

    // The same for the last element of the block, the other value that we're going to sum up
    if (start + BLOCK_SIZE + ty < N)
       partialSum[BLOCK_SIZE + ty + column] = powf(input[ (start + BLOCK_SIZE + ty)*MAXN + x ] - means [ x ], 2)
    else
       partialSum[BLOCK_SIZE + ty + column] = 0;  

    // Perform the partial sum
    for (unsigned int stride = BLOCK_SIZE; stride >= 1; stride >>= 1) {
       __syncthreads();
       if (ty < stride)
          partialSum[ty + column] += partialSum[ty+stride + column];
    }
    // After the loop, the partial sum is found in partialSum[0]
    // So we have to put it in the output array
    if (ty == 0)
       output[blockIdx.y*N + x] = partialSum[column];
}



__global__ void calculateQuadratic(float * input, float * means, const int N) {

    unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;

    // Verify that we are inside the array, so CUDA won't throw errors
    if ( y >= N || x >= N )
      return;

    input[ x + y*MAXN ] = powf(input[ x + y*MAXN ] - means [ x ], 2);
}



void matrixNorm() {

  printf("Computing using CUDA.\n");

  // CALCULATING MEAN
  int size = MAXN*MAXN*sizeof(float);
  int Nsums = ceil( ((float)N) / (BLOCK_SIZE<<1));
  int sizeSums = N*Nsums*sizeof(float);
/*
  int Nsums2 = ceil( ((float)Nsums) / (BLOCK_SIZE<<1));
  int sizeSums2 = N*Nsums2*sizeof(float);
*/
  int row, col;

  float *d_sums, *d_A, *d_B;

  //Get user input into size;
  //float (*h_sums)[BLOCK_SIZE] = new float[N][BLOCK_SIZE];
  float *h_sums;
  h_sums = (float*)malloc(sizeSums);
  for (int i=0; i < Nsums; i++)
      for (int j=0; j < N; j++)
          h_sums[i*N + j] = -1;
      

  printf("MATRIX h_sums BEFORE\n\t");
  for (row = 0; row < Nsums; row++) {
      for (col = 0; col < N; col++) {
          printf("%1.1f%s", h_sums[row*N + col], (col < N-1) ? ", " : ";\n\t");
      }
  }

  for (int i=0; i < N; i++)
      for (int j=0; j < N; j++) {
        if ( i == 0 )
          A[i][j] = j;
        else
          A[i][j] = 1;
      }

  printf("MATRIX A BEFORE\n\t");
  for (row = 0; row < N; row++) {
      for (col = 0; col < N; col++) {
          printf("%1.1f%s", A[row][col], (col < N-1) ? ", " : ";\n\t");
      }
  }
  

  // Allocagte space for variables
  printError( hipMalloc( (void**)&d_A, size ) );
  printError( hipMalloc( (void**)&d_B, size ) );
  printError( hipMalloc( (void**)&d_sums, sizeSums ) );

  printError( hipMemcpy( d_A, A, size, hipMemcpyHostToDevice) );
  printError( hipMemcpy( d_sums, h_sums, sizeSums, hipMemcpyHostToDevice ));

  int gridSize = ceil(((float)N)/BLOCK_SIZE);

  dim3 dimBlock( BLOCK_SIZE, BLOCK_SIZE );
  dim3 dimGrid( gridSize, gridSize);

  // 
  // Use reduction with partial sum algorithm to create partial sums of column values with complexity O(log(N))
  //
  partialSum<<< dimGrid, dimBlock>>> (d_A, d_sums, N);

  printError( hipMemcpy( h_sums, d_sums, sizeSums, hipMemcpyDeviceToHost ) );

  // 
  // Add reducted means sequentially. After that, divide by N, total number of elements in a column
  //
  float *h_means;
  h_means = (float*)malloc( N*sizeof(float) );
  for ( int i = 0; i < N; i++ )
    h_means[i] = 0;

  for ( int i = 0; i < Nsums; i++ )
    for ( int j = 0; j < N; j++ )
      h_means[j] += h_sums[i*N+j];

  // Divide between number of elements
  for ( int i = 0; i < N; i++ )
    h_means[i] /= N;

  printf("MATRIX h_means AFTER\n\t");
  for ( int i = 0; i < N; i++ )
    printf("%1.2f%s", h_means[i], (i < N-1) ? ", " : ";\n\t");


  // 
  // Transfer means to CUDA
  //
  float *d_means;
  printError( hipMalloc( (void**)&d_means, N*sizeof(float) ) );
  printError( hipMemcpy( d_means, h_means, N*sizeof(float), hipMemcpyHostToDevice) );

  // 
  // Calculate the value of (A[i][j] - mean)^2
  // We put the value in the input array because we won't need the original a[i][j] values anymore
  //
  /*
  calculateQuadratic<<< dimGrid, dimBlock>>> (d_A, d_means, N);

  printf("MATRIX A AFTER QUADRATIC CALCULATION\n\t");
  for (row = 0; row < N; row++) {
      for (col = 0; col < N; col++) {
          printf("%1.3f%s", A[row][col], (col < N-1) ? ", " : ";\n\t");
      }
  }*/

  //
  // Add all the operands (A[i][j] - mean)^2 in each column
  // It is the same operation of adding all values in columns that we did in the step of calculating the mean
  //
  partialSum2<<< dimGrid, dimBlock>>> (d_A, d_sums, d_means, N);

  printError( hipMemcpy( A, d_A, size, hipMemcpyDeviceToHost ) );

  printError( hipMemcpy( h_sums, d_sums, sizeSums, hipMemcpyDeviceToHost ) );

  // 
  // Add reducted means sequentially. After that, divide by N and calculate square root
  //
  for ( int i = 0; i < N; i++ )
    h_means[i] = 0;

  for ( int i = 0; i < Nsums; i++ )
    for ( int j = 0; j < N; j++ )
      h_means[j] += h_sums[i*N+j];

  // Divide between number of elements
  for ( int i = 0; i < N; i++ )
    h_means[i] = powf(h_means[i]/N, 0.5f);

  printf("MATRIX h_means AFTER QUADRATIC ADDING\n\t");
  for ( int i = 0; i < N; i++ )
    printf("%1.2f%s", h_means[i], (i < N-1) ? ", " : ";\n\t");

  
  // 
  // Apply the formula to normalize
  // B[row][col] = (A[row][col] – mean) / standard_deviation
  //


  printError( hipFree(d_A) );
  printError( hipFree(d_B) );
  printError( hipFree(d_means) );
  
}


