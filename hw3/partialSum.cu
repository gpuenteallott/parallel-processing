#include "hip/hip_runtime.h"


#include <stdio.h>
#include <iostream>

using namespace std;

#define BLOCK_SIZE 8

__global__ void 
partialSum(float *input, float *output, const int N) {

	//@@ Load a segment of the input vector into shared memory
    __shared__ float partialSum[2 * BLOCK_SIZE];
    unsigned int t = threadIdx.x, start = 2 * blockIdx.x * BLOCK_SIZE;

    if (start + t < N)
       partialSum[t] = input[start + t];
    else
       partialSum[t] = 0;
   
    if (start + BLOCK_SIZE + t < N)
       partialSum[BLOCK_SIZE + t] = input[start + BLOCK_SIZE + t];
    else
       partialSum[BLOCK_SIZE + t] = 0;
   
    //@@ Traverse the reduction tree
    for (unsigned int stride = BLOCK_SIZE; stride >= 1; stride >>= 1) {
       __syncthreads();
       if (t < stride)
          partialSum[t] += partialSum[t+stride];
    }

    //@@ Write the computed sum of the block to the output vector at the 
    //@@ correct index
    if (t == 0)
       output[blockIdx.x] = partialSum[0];

}

__global__ void total(float * input, float * output, int len) {

    //@@ Load a segment of the input vector into shared memory
    __shared__ float partialSum[2 * BLOCK_SIZE];
    unsigned int t = threadIdx.x, start = 2 * blockIdx.x * BLOCK_SIZE;
    if (start + t < len)
       partialSum[t] = input[start + t];
    else
       partialSum[t] = 0;
    if (start + BLOCK_SIZE + t < len)
       partialSum[BLOCK_SIZE + t] = input[start + BLOCK_SIZE + t];
    else
       partialSum[BLOCK_SIZE + t] = 0;
    //@@ Traverse the reduction tree
    for (unsigned int stride = BLOCK_SIZE; stride >= 1; stride >>= 1) {
       __syncthreads();
       if (t < stride)
          partialSum[t] += partialSum[t+stride];
    }
    //@@ Write the computed sum of the block to the output vector at the 
    //@@ correct index
    if (t == 0)
       output[blockIdx.x] = partialSum[0];
}


int
main()
{   
	int N = 25;
	int sizeInput = N*sizeof(float);
	int sizeOutput = ceil( ((float)N) / (BLOCK_SIZE<<1) *sizeof(float);

	float *d_a, *h_a, *h_o, *d_o;

	h_a = (float*)malloc(num_bytes);
	h_o = (float*)malloc(sizeOutput);

	for (int i=0; i < N; i++){   
	    h_a[i]=0; h_o[i]=0;
	}

	h_a[0]=1; 
	h_a[1]=1;
	h_a[2]=1;
	h_a[3]=1;
	h_a[4]=1;
	h_a[5]=1;
	h_a[6]=1;
	h_a[7]=1;
	h_a[8]=1;
	h_a[9]=1;
	h_a[10]=1;
	h_a[11]=1;
	h_a[12]=1;
	h_a[13]=1;
	h_a[14]=1;
	h_a[15]=1;
	h_a[16]=1;
	h_a[17]=1;
	h_a[18]=1;
	h_a[19]=1;
	h_a[20]=1;
	h_a[21]=1;
	h_a[22]=1;
	h_a[23]=1;
	h_a[24]=1;

	printf("MATRIX BEFORE\n\t");
    int i;
	for (i = 0; i < N; i++) {
      cout << "h_a[" << i << "]=" << h_a[i] << endl;
    } 

	hipMalloc( (void**)&d_a, sizeInput );
	hipMalloc( (void**)&d_o, sizeOutput );
	hipMemcpy( d_a, h_a, sizeInput, hipMemcpyHostToDevice);
	hipMemcpy( d_o, h_o, sizeOutput, hipMemcpyHostToDevice);

	dim3 dimBlock( BLOCK_SIZE, 1 );
	dim3 dimGrid( ceil(  ((float)N)/BLOCK_SIZE), 1 );

	partialSum<<< dimGrid, BLOCK_SIZE>>> (d_a, d_o, N);

	hipMemcpy( h_a, d_a, sizeInput, hipMemcpyDeviceToHost );
	hipMemcpy( h_o, d_o, sizeOutput, hipMemcpyDeviceToHost );

	hipFree(d_a);
	hipFree(d_o);

	printf("MATRIX AFTER\n\t");
	for (i = 0; i < N; i++) {
      cout << "h_o[" << i << "]=" << h_o[i] << endl;
    } 
    free(h_a);
    free(h_o);
}
