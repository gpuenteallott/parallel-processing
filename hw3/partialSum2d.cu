


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <iostream>

using namespace std;

#define BLOCK_SIZE 8

// http://stackoverflow.com/questions/20086047/cuda-matrix-example-block-size
void printError(hipError_t err) {
    if(err != 0) {
        printf("CUDA ERROR: %s\n", hipGetErrorString(err));
        getchar();
    }
}



/**
This function performs the partial sum of the given arrays
It is an improvement over the partial sum example from class
Inspired in the code found in https://gist.github.com/wh5a/4424992
The code there has been studied, as the comments indicate
*/
__global__ void 
partialSum(float *input, float *output, const int N, const int Noutput) {

	// Load a segment of the input vector into shared memory
	// This is because the entire array might be too big and is stored into the global memory
    __shared__ float partialSum[2* BLOCK_SIZE*BLOCK_SIZE];

    // Position in the input array
    unsigned int t = threadIdx.x;
    unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned int ty = threadIdx.y;
    unsigned int tx = threadIdx.x;


    if ( y >= N )
      return;

    // Start is the beining of the current calculations
    // If blockIdx is not 0, then the result will go to the blockIdx position of the output array
    unsigned int start = 2 * blockIdx.y * BLOCK_SIZE;

    // If we are inside the input array, we transfer the value that we're going to sum up to the partial sum array
    if (start + t < N)
       //partialSum[t+ty*BLOCK_SIZE] = input[start + t +y*N];
      partialSum[t + tx*2*BLOCK_SIZE] = input[start + t + x*N];
    else
       //partialSum[t+ty*BLOCK_SIZE] = 0;
      partialSum[t + tx*2*BLOCK_SIZE] = 0;
   
    // The same for the last element of the block, the other value that we're going to sum up
    if (start + BLOCK_SIZE + t < N)
       //partialSum[BLOCK_SIZE + t+ty*BLOCK_SIZE] = input[start + BLOCK_SIZE + t +y*N];
      partialSum[BLOCK_SIZE + t + tx*2*BLOCK_SIZE] = input[start + BLOCK_SIZE + t + x*N];
    else
       //partialSum[BLOCK_SIZE + t+y*2*BLOCK_SIZE] = 0;
      partialSum[BLOCK_SIZE + t + tx*2*BLOCK_SIZE] = 0;
   
    // Perform the partial sum
    for (unsigned int stride = BLOCK_SIZE; stride >= 1; stride >>= 1) {
       __syncthreads();
       if (t < stride)
          //partialSum[t+ty*BLOCK_SIZE] += partialSum[t+stride+ty*BLOCK_SIZE];
          partialSum[t + tx*2*BLOCK_SIZE] += partialSum[t+stride + tx*2*BLOCK_SIZE];
    }

    // After the loop, the partial sum is found in partialSum[0]
    // So we have to put it in the output array
    if (t == 0)
       //output[blockIdx.x + y*Noutput] += partialSum[0+ty*BLOCK_SIZE];
      output[blockIdx.x + y*Noutput] = partialSum[tx*2*BLOCK_SIZE];
}


int
main()
{   
	int N = 24;
	int sizeInput = N*N*sizeof(float);
	int Noutput = ceil( ((float)N) / (BLOCK_SIZE<<1));
	int sizeOutput = N*Noutput*sizeof(float);
  int row, col;

  float h_a[N][N];
  float h_o[N][Noutput];
  for(int i = 0; i < N; i++) {
    for(int j = 0; j < N; j++) {
        h_a[i][j] = i;
        h_o[i][j] = 0;
    }
  }

  float (*d_A)[N]; //pointers to arrays of dimension N
  float (*d_O)[N];

  float *d_a, *d_o;
  /*
	float *d_a, *h_a, *h_o, *d_o;
*//*
	h_a = (float*)malloc(sizeInput);
	h_o = (float*)malloc(sizeOutput);
*/
  /*
	for (int i=0; i < N; i++)
      for (int j=0; j < N; j++)
	       h_a[i*N+j]=i+1;
  for (int i=0; i < Noutput; i++)
      for (int j=0; j < N; j++)
         h_o[i*Noutput+j]=0;
	*/

  printf("MATRIX O BEFORE\n\t");
  for (row = 0; row < Noutput; row++)
    for (col=0; col < N; col++)
      printf("%1.1f%s", h_o[row][col], (col < N-1) ? ", " : ";\n\t");


	printf("MATRIX A BEFORE\n\t");
	for (row = 0; row < N; row++)
    for (col=0; col < N; col++)
      printf("%1.1f%s", h_a[row][col], (col < N-1) ? ", " : ";\n\t");

	printError( hipMalloc( (void**)&d_a, sizeInput ) );
	printError( hipMalloc( (void**)&d_o, sizeOutput ) );
	printError( hipMemcpy( d_a, h_a, sizeInput, hipMemcpyHostToDevice) );
	printError( hipMemcpy( d_o, h_o, sizeOutput, hipMemcpyHostToDevice) );

	dim3 dimBlock( BLOCK_SIZE, BLOCK_SIZE );
	dim3 dimGrid( ceil(  ((float)N)/BLOCK_SIZE), ceil(  ((float)N)/BLOCK_SIZE) );

	partialSum<<< dimGrid, dimBlock>>> (d_a, d_o, N, Noutput);

	printError( hipMemcpy( h_a, d_a, sizeInput, hipMemcpyDeviceToHost ) );
	printError( hipMemcpy( h_o, d_o, sizeOutput, hipMemcpyDeviceToHost ) );

	printError( hipFree(d_a) );
	printError( hipFree(d_o) );

	printf("MATRIX AFTER\n\t");
	for (row = 0; row < Noutput; row++)
    for (col=0; col < N; col++)
      printf("%1.1f%s", h_o[row][col], (col < N-1) ? ", " : ";\n\t");


}


